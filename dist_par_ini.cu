// ----------------------------------------------------------------------------
// Distância de edição paralelizado (solução inicial)
//
// Estudante: Andrews Matheus de Oliveira
// Estudante: Guilherme Gonzaga de Andrade
// Estudante: Walter do Espirito Santo Souza Filho
//
// Para compilar: nvcc dist_par_ini.cu -o dist_par_ini
// Para executar: ./dist_par_ini <nome arquivo entrada>
// ----------------------------------------------------------------------------



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


char *aloca_sequencia_host(int n) {
	char *seq = (char *) malloc((n + 1) * sizeof(char));

	if (seq == NULL) {
		printf("\nErro na alocação de estruturas\n");
		exit(EXIT_FAILURE);
	}

	return seq;
}


void ler_entrada(const char *nomearq, int *n, int *m, char **h_s, char **h_r) {
	// Abre arquivo de entrada
	FILE *arqEntrada = fopen(nomearq, "rt");

	if (arqEntrada == NULL) {
		printf("\nArquivo texto de entrada não encontrado\n");
		exit(EXIT_FAILURE);
	}

	// Lê tamanho das sequências s e r
	fscanf(arqEntrada, "%d %d", n, m);

	*h_s = aloca_sequencia_host(*n);
	*h_r = aloca_sequencia_host(*m);

	(*h_s)[0] = ' ';
	(*h_r)[0] = ' ';
	fscanf(arqEntrada, "%s", *h_s + 1);
	fscanf(arqEntrada, "%s", *h_r + 1);

	// Fecha arquivo de entrada
	fclose(arqEntrada);
}


void aloca_dev(int n, int m, char **d_s, char **d_r, int **d_dist) {
	if (hipSuccess != hipMalloc(d_s, (n + 1) * sizeof(char)) ||
	    hipSuccess != hipMalloc(d_r, (m + 1) * sizeof(char)) ||
	    hipSuccess != hipMalloc(d_dist, (n + 1) * (m + 1) * sizeof(int))) {
		printf("\nErro na alocação de estruturas\n");
		exit(EXIT_FAILURE);
	}
}


__device__
int min(int a, int b, int c) {
	if (a < b)
		return (a < c) ? a : c;
	else
		return (b < c) ? b : c;
}


__device__
void inicializa_matriz(int n, int m, int *mat) {
	mat[(m + 1) * (threadIdx.x + 1)] = threadIdx.x + 1;

	for (int i = threadIdx.x; i <= m; i += blockDim.x)
		mat[i] = i;
}

__global__
void distancia_edicao_adiagonal(int n, int m, const char *d_s, const char *d_r, int *d_dist) {
	inicializa_matriz(n, m, d_dist);

	// Para cada anti-diagonal
	for (int aD = 2; aD <= n + m; aD++) {
		// Calcula índices i e j da célula (linha e coluna)
		int i = n - threadIdx.x;
		int j = aD - i;

		// Se é uma célula válida
		if (1 <= j && j <= m) {
			int a = d_dist[(m+1) * i + j-1] + 1;
			int b = d_dist[(m+1) * (i-1) + j] + 1;
			int c = d_dist[(m+1) * (i-1) + j-1] + (d_s[i] != d_r[j]);
			d_dist[(m+1) * i + j] = min(a, b, c);
		}
		__syncthreads();
	}
}


int main(int argc, const char **argv) {
	int n, m;  // Tamanho das sequências s e r
	char *h_s;  // Sequência s de entrada (com tamanho n+1)
	char *h_r;  // Sequência r de entrada (com tamanho m+1)
	int *d_dist;  // Matriz de distâncias com tamanho (n+1)*(m+1)
	char *d_s, *d_r;  // Cópias das sequências no device

	if (argc != 2) {
		printf("O programa foi executado com argumentos incorretos.\n");
		printf("Uso: ./dist_seq <nome arquivo entrada>\n");
		return EXIT_FAILURE;
	}

	// Lê sequências do arquivo de entrada
	ler_entrada(argv[1], &n, &m, &h_s, &h_r);

	// Aloca estruturas no device
	aloca_dev(n, m, &d_s, &d_r, &d_dist);

	float tempo = 0;  // Tempo de execução na CPU em milissegundos
	hipEvent_t d_ini, d_fim;
	hipEventCreate(&d_ini);
	hipEventCreate(&d_fim);
	hipEventRecord(d_ini, 0);

	hipMemcpy(d_s, h_s, (n + 1) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_r, h_r, (m + 1) * sizeof(char), hipMemcpyHostToDevice);

	int nBlocos = 1, nThreadsBloco = n;

	// Calcula distância de edição entre sequências s e r, por anti-diagonais
	distancia_edicao_adiagonal<<<nBlocos, nThreadsBloco>>>(n, m, d_s, d_r, d_dist);
	hipDeviceSynchronize();

	int dist_total = -1;
	hipMemcpy(&dist_total, d_dist + (n+1) * (m+1) - 1, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", dist_total);

	hipEventRecord(d_fim, 0);
	hipEventSynchronize(d_fim);
	hipEventElapsedTime(&tempo, d_ini, d_fim);
	hipEventDestroy(d_ini);
	hipEventDestroy(d_fim);

	printf("%.2f\n", tempo);

	// Libera vetores s e r e matriz de distâncias
	free(h_s);
	free(h_r);
	hipFree(d_s);
	hipFree(d_s);
	hipFree(d_dist);

	return EXIT_SUCCESS;
}
